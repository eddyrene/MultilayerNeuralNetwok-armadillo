
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
using namespace std;

#define Mask_width  3
#define Mask_radius Mask_width/2
#define TILE_WIDTH 32
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0), 255))

__global__ 
void convolution(double *I, const int* __restrict__ M, double *P, int channels, int width, int height) 
{
   __shared__ double N_ds[w][w];
   int k;
   for (k = 0; k < channels; k++) {
      // First batch loading
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
         destY = dest / w, destX = dest % w,
         srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius,
         srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
         src = (srcY * width + srcX) * channels + k;
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         N_ds[destY][destX] = I[src];
      else
         N_ds[destY][destX] = 0;

      // Second batch loading
      dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
      destY = dest / w, destX = dest % w;
      srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
      srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
      src = (srcY * width + srcX) * channels + k;
      if (destY < w) {
         if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = I[src];
         else
            N_ds[destY][destX] = 0;
      }
      __syncthreads();

      int accum = 0;
      int y, x;
      for (y = 0; y < Mask_width; y++)
         for (x = 0; x < Mask_width; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
      y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         P[(y * width + x) * channels + k] = clamp(accum);
      __syncthreads();
   }
}

void WritePGM(double * Pout, int fil , int cols,const  char *name)
{
    ofstream file(name);
    file<<"P2"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<n)
    {
        file<<(int)Pout[i]<<endl;
        i++;
    }
}
void kernel_convolution(double *R , double *sR , int order ,int chanel )
{
   double * d_R;//,*d_G,*d_B;
   double * sd_R;//,*sd_G,*sd_B;
   int N=order;
   int M=order;

   int THREADS_PER_BLOCK = 32;
   int size =1*N*M*sizeof(double);
    //cout<<"tamano Imagen "<<N<<" "<<M<<"  size "<<size<<endl;
   int k[9]={-1,0,1,-2,0,2,-1,0,1};
   int *d_k;

   hipMalloc((void **)&d_R, size);
   hipMalloc((void **)&sd_R, size);
   hipMalloc((void **)&d_k,9*sizeof(int));   
   
   hipMemcpy(d_R, R, size, hipMemcpyHostToDevice);
   hipMemcpy(d_k, k, 9*sizeof(int), hipMemcpyHostToDevice);

   int blocks= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
   dim3 dimGrid(blocks, blocks, 1);
   dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, 1);
      convolution<<<dimGrid,dimBlock>>>(d_R,d_k,sd_R,chanel, N, M);
   hipMemcpy(sR, sd_R, size, hipMemcpyDeviceToHost);
   string name = "result.ppm";
   WritePGM(sR, N,M,name.c_str());  
   //free(R); //free(G);free(B);
   hipFree(d_R); //cudaFree(d_B);cudaFree(d_G);
   hipFree(sd_R); //cudaFree(sd_B);cudaFree(sd_G);
   //return 0;
}