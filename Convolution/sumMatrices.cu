#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <cmath>
#include <string>
using namespace std;

//#define THREADS_PER_BLOCK 32

void fillMatrix(int* a, int n)
{
   int i;
   for (i = 0; i < n*n; ++i)
        a[i] = 10;//rand()%5;
}

__global__ 
void matrixAdition(int *c, int *a, int *b,int n) 
{
    int ij = threadIdx.x + blockDim.x * blockIdx.x;
		if(ij<(n*n))
			c[ij] = a[ij] + b[ij];
}

__global__ 
void matrixAditionRow(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
   //	if(blockDim.x != 0)
   	//printf("%d  salida\n", ij);
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij*n+i] = a[ij*n+i] + b[ij*n+i];
	}
}

__global__ void convolution_1D_basic_kernel(float *N, float *M, float *P,int Mask_Width , int Width)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	float Pvalue = 0;
	int N_start_point = i - (Mask_Width/2);
	for (int j = 0; j < Mask_Width; j++) 
	{
		if (N_start_point + j >= 0 && N_start_point + j < Width) 
		{
			Pvalue += N[N_start_point + j]*M[j];
		}
	}
	P[i] = Pvalue;
}

__global__ 
void matrixAditionCol(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij+n*i] = a[ij+n*i] + b[ij+n*i];
	}
}

void printMatrix(string s, int *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		for(int j=0;j<tam;j++)
		{
			cout<<a[i*tam+j]<<" ";
		}
		cout<<endl;
	}
}

void ReadPPM(int *Pin, char *name)
{
	int e1;
    string line,s1;
    ifstream file(name);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    int m=0;
    while(!file.eof())
    {
        file>>e1;
       // cout<<e1<<endl;
        //if(!e1) break;
        Pin[m]=e1;
        m++;
    }
}

void WritePGM(int * Pout, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P2"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<n)
    {
        file<<Pout[i]<<endl;
        i++;
    }
}

void WritePPM(int * Pout, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P3"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<3*n)
    {
        file<<Pout[i]<<endl;
        i++;
    }
}

int main(int argc, char *argv[])
{
	srand (time(NULL));
	int  N= strtol(argv[1], NULL, 10);
		//matrixAditionCol<<<blocks2,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Elapsed time : %f ms\n" ,elapsedTime);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//printMatrix("Printing Matrix A \n",a,N);
	//printMatrix("Printing Matrix B \n",b,N);
	//printMatrix("Printing Matrix C \n",c,N);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
