
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <cmath>
#include <string>
using namespace std;

//#define THREADS_PER_BLOCK 32

#define Mask_width  3
#define Mask_radius Mask_width/2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0), 255))



void fillMatrix(int* a, int n)
{
   int i;
   for (i = 0; i < n*n; ++i)
        a[i] = 10;//rand()%5;
}

__global__ 
void matrixAdition(int *c, int *a, int *b,int n) 
{
    int ij = threadIdx.x + blockDim.x * blockIdx.x;
		if(ij<(n*n))
			c[ij] = a[ij] + b[ij];
}

__global__ 
void matrixAditionRow(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
   //	if(blockDim.x != 0)
   	//printf("%d  salida\n", ij);
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij*n+i] = a[ij*n+i] + b[ij*n+i];
	}
}

__global__ 
void convolution_1D_basic_kernel(int *R, int *G, int *B , int *M, int *sd_R, int *sd_G, int *sd_B, int Mask_Width , int Width)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int r = 0;
	int g = 0;
	int b = 0;
	int N_start_point = i - (Mask_Width/2);
	for (int j = 0; j < Mask_Width; j++) 
	{
		if (N_start_point + j >= 0 && N_start_point + j < Width) 
		{
			r += R[N_start_point + j]*M[j];
			g += G[N_start_point + j]*M[j];
			b += B[N_start_point + j]*M[j];
		}
	}
	sd_R[i] = r;
	sd_G[i] = g;
	sd_B[i] = b;
}

__global__ 
void convolution(int *I, const int* __restrict__ M, int *P, int channels, int width, int height) 
{
   __shared__ int N_ds[w][w];
   int k;
   for (k = 0; k < channels; k++) {
      // First batch loading
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
         destY = dest / w, destX = dest % w,
         srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius,
         srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
         src = (srcY * width + srcX) * channels + k;
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         N_ds[destY][destX] = I[src];
      else
         N_ds[destY][destX] = 0;

      // Second batch loading
      dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
      destY = dest / w, destX = dest % w;
      srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
      srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
      src = (srcY * width + srcX) * channels + k;
      if (destY < w) {
         if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = I[src];
         else
            N_ds[destY][destX] = 0;
      }
      __syncthreads();

      int accum = 0;
      int y, x;
      for (y = 0; y < Mask_width; y++)
         for (x = 0; x < Mask_width; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
      y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         P[(y * width + x) * channels + k] = clamp(accum);
      __syncthreads();
   }
}
/*
#define P2D(PTR, PITCH, ROW, COL, TYPE)    ((TYPE *)( (char *)(PTR) + (ROW) * (PITCH) ) )[(COL)]

__global__ void convolutionKernel(int *inImg, int *outImg, ROI size, int pitch, int *mask, size_t maskPitch, int maskSize, int maskSum)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row > size.height || col > size.width)
        return;

    int k = maskSize / 2;
    
    int pixelNewValue = 0;
    int p;
    int m;

    // Convolution
    for (int i = -k; i <= k; i++)
    {
        for (int j = -k; j <= k; j++)
        {
            // Pixel
            p = P2D(inImg, pitch, (row + i), (col + j), int);
            
            // Mask
            m = P2D(mask, maskPitch, (i+k), (j+k), int);
            
            pixelNewValue += m * p;
        }
    }

    // New value
    pixelNewValue /= maskSum;
    if (pixelNewValue < 0) pixelNewValue = 0;
    else if (pixelNewValue > 255) pixelNewValue = 255;
 
    // Set value of pixel
    P2D(outImg, pitch, row, col, int) = pixelNewValue;
} 
*/

__global__ 
void matrixAditionCol(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij+n*i] = a[ij+n*i] + b[ij+n*i];
	}
}

void printMatrix(string s, int *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		for(int j=0;j<tam;j++)
		{
			cout<<a[i*tam+j]<<" ";
		}
		cout<<endl;
	}
}

void ReadPPM(int *Pin, char *name)
{
	int e1;
    string line,s1;
    ifstream file(name);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    int m=0;
    while(!file.eof())
    {
        file>>e1;
    //    cout<<e1<<endl;
        //if(!e1) break;
        Pin[m]=e1;
        m++;
    }
}

void ReadPPM(int *R,int *G , int *B, char *name)
{
	int e1;
    string line,s1;
    ifstream file(name);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    int m=0;
    while(!file.eof())
    {
        file>>e1;
        R[m]=e1;
        file>>e1;
        G[m]=e1;
        file>>e1;
        B[m]=e1;
        m++;
    }
}
int* ReadSizeImg(char * name)
{
    int * dim= new int[2];
    int fil, col;
    string line,s1;
    ifstream file(name);
    getline(file,line);
    getline(file,line);
    file>>fil>>col;
    dim[0]=fil; dim[1]=col;   
    return dim;
}

void WritePGM(int * Pout, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P2"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<n)
    {
        file<<Pout[i]<<endl;
        i++;
    }
}

void WritePPM(int * Pout, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P3"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<3*n)
    {
        file<<Pout[i]<<endl;
        i++;
    }
}

void WritePPM(int * R, int* G,int *B, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P3"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<n)
    {
        file<<R[i]<<endl;
        file<<G[i]<<endl;
        file<<B[i]<<endl;
        i++;
    }
}
void print_vect(int *V, int n){
    int i;
    for (i = 0; i < n; i++)
		printf("%d ", V[i]);
}
int main(int argc, char *argv[])
{
	int * R;//,*G,*B;
	int * sR;//,*sG,*sB;
    int * d_R;//,*d_G,*d_B;
    int * sd_R;//,*sd_G,*sd_B;
	int * order = ReadSizeImg("img.pgm");
	int N=order[0]; int M=order[1];

	int THREADS_PER_BLOCK = 16;
	int size =3*N*M*sizeof(int);

    cout<<"tamano Imagen "<<N<<" "<<M<<"  size "<<size<<endl;

    int k[9]={-1,0,1,-2,0,2,-1,0,1};
	int *d_k;

    hipMalloc((void **)&d_R, size);
	//cudaMalloc((void **)&d_G, size);
	//cudaMalloc((void **)&d_B, size);
	hipMalloc((void **)&sd_R, size);
	//cudaMalloc((void **)&sd_G, size);
	//cudaMalloc((void **)&sd_B, size);
	hipMalloc((void **)&d_k,9*sizeof(int));

   
    R = (int *)malloc(size);
   // G = (int *)malloc(size); 
    //B = (int *)malloc(size);
    ReadPPM(R,"img.pgm");
     cout<<"pasa"<<endl;
       // ;printf("\n Impriendo R \n");
    	//print_vect(R,order[0]*order[1]); printf("\nImpriendo B \n");
    	//print_vect(G,order[0]*order[1]);printf("\nImpriendo G \n");
    	//print_vect(B,order[0]*order[1]);
    sR = (int *)malloc(size); 
   // sG = (int *)malloc(size); 
    //sB = (int *)malloc(size); 

	//for(int i=0;i<N*N;i++)
	//	sR[i]=0;
	//print_vect(sR,order[0]*order[1]);
	hipMemcpy(d_R, R, size, hipMemcpyHostToDevice);
    //cudaMemcpy(d_G, G, size, cudaMemcpyHostToDevice);
	//cudaMemcpy(d_B, B, size, cudaMemcpyHostToDevice);
	hipMemcpy(d_k, k, 9*sizeof(int), hipMemcpyHostToDevice);




	int blocks= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	dim3 dimGrid(blocks, blocks, 1);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, 1);
	
	cout<<"blocks : \n"<<blocks<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; 
	convolution<<<dimGrid,dimBlock>>>(d_R, d_k ,sd_R,1, N, M);
    //convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,imageChannels, imageWidth, imageHeight);

      // convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
                                   //   imageChannels, imageWidth, imageHeight);
		//blurKernel<<<dimGrid,dimBlock>>>( d_Pout, d_Pin, N, M);
	hipMemcpy(sR, sd_R, size, hipMemcpyDeviceToHost);
	//cudaMemcpy(sG, sd_G, size, cudaMemcpyDeviceToHost);
	//cudaMemcpy(sB, sd_B, size, cudaMemcpyDeviceToHost);

	//printf("\n Impriendo R \n");
	//print_vect(sR,order[0]*order[1]); printf("\nImpriendo B \n");
	//print_vect(sG,order[0]*order[1]);printf("\nImpriendo G \n");
	//print_vect(sB,order[0]*order[1]);
	//WritePPM(sR,sG,sB,N,M,"convLena.ppm");  
    cout<<"ss"<<endl;
	WritePPM(sR, N,M,"siete.ppm");  
	free(R); //free(G);free(B);
	hipFree(d_R); //cudaFree(d_B);cudaFree(d_G);
	hipFree(sd_R); //cudaFree(sd_B);cudaFree(sd_G);
	return 0;
}
